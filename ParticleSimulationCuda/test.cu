#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <glm/glm.hpp>

//hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);


__global__ void addArray(float* A, float* B, float* C, int numElements) {
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i] + 0.f;
    }
}
extern "C" float routine(float* A, float* B, float* C, int numElements) {
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);

    float* d_A = NULL;
    err = hipMalloc((void**)&d_A, size );

    float* d_B = NULL;
    err = hipMalloc((void**)&d_B, size );

    float* d_C = NULL;
    err = hipMalloc((void**)&d_C, size );

    err = hipMemcpy(d_A, A, size , hipMemcpyHostToDevice);
    err = hipMemcpy(d_B, B, size , hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    addArray <<<blocksPerGrid, threadsPerBlock >>> (d_A, d_B, d_C, numElements);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return err;
}


